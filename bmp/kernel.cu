#include "hip/hip_runtime.h"
﻿#include "bmp.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <omp.h>
void printDeviceProp()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	printf("Device name : %s\n", deviceProp.name);
	printf("Total global memory : %d MB\n",
		deviceProp.totalGlobalMem / 1024 / 1024);
	printf("Shared memory per block : %d\n",
		deviceProp.sharedMemPerBlock);
	printf("Registers per block : %d\n",
		deviceProp.regsPerBlock);
	printf("Warp size : %d\n", deviceProp.warpSize);
	printf("Memory pitch : %d\n", deviceProp.memPitch);
	printf("Max threads per block : %d\n",
		deviceProp.maxThreadsPerBlock);
	printf("Max threads dimensions : x = %d, y = %d, z =% d\n", deviceProp.maxThreadsDim[0],
		deviceProp.maxThreadsDim[1],
		deviceProp.maxThreadsDim[2]);
	printf("Max grid size: x = %d, y = %d, z = %d\n",
		deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
		deviceProp.maxGridSize[2]);
	printf("Clock rate: %d\n", deviceProp.clockRate);
	printf("Total constant memory: %d\n",
		deviceProp.totalConstMem);
	printf("Compute capability: %d.%d\n",
		deviceProp.major, deviceProp.minor);
	printf("Texture alignment: %d\n",
		deviceProp.textureAlignment);
	printf("Device overlap: %d\n",
		deviceProp.deviceOverlap);
	printf("Multiprocessor count: %d\n",
		deviceProp.multiProcessorCount);
	printf("Kernel execution timeout enabled: %s\n", deviceProp.kernelExecTimeoutEnabled ? "true" :
		"false");
	scanf("");
}


__constant__ int color_palette[256];




/*
INPUT: COLORS CONVERTED TO INT FROM GPU, POINTER TO RESULTING PALETTE, SIZE OF COLORS ARRAY, NUM OF COLORS THAT EACH THREAD PROCEED
OUTPUT: COLOR PALETTE
the task boils down to finding unique numbers in an unsorted array
each value from colors array inserted to hash table by index: int COLOR * 0xDEADBEEF >> 19 
if value in hash table by index is busy, thread tries to find new index: index = (index + 1) & 4095;
if this hash table's value busy too, thread tries next indexes
*/

__global__ void d_createColorPalette(int* d_all_colors, int* d_color_palette, size_t size,int* temp_colors)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	if (ix < size)
	{
		int index = (d_all_colors[ix] * 0xDEADBEEF) >> 19;
				while (true)
				{
					int prev = atomicCAS(&temp_colors[index], 0, d_all_colors[ix]);
					if (prev == 0 || prev == d_all_colors[ix])
					{
						temp_colors[index] = d_all_colors[ix];
						break;
					}
					index = (index + 1) & 9999;
				}
	}
}
__global__ void writePalette(int* d_color_palette, size_t size, int* temp_colors, int* num)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	if (ix < 10000)
	{
		if (temp_colors[ix] != 0)
		{
			d_color_palette[atomicAdd(num, 1)] = temp_colors[ix];
		}
	}
}
/*
INPUT: COLORS CONVERTED TO INT FROM GPU, COLOR PALETTE, RESULTING UINT8 ARRAY, SIZE OF COLORS ARRAY
OUTPUT: UINT8 COLORS ARRAY
EACH THREAD PROCESSES ONLY ONE COLOR:TRIES TO FIND COLOR IN COLOR PALETTE AND WRITES COLOR'S NUMBER TO RESULTING ARRAY
*/
__global__ void d_applyPalette(int* d_all_colors, int* d_color_palette, UINT8* d_result, size_t size)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	if (ix < size)
		for (int i = 0; i < 256; i++)
			if (d_all_colors[ix] == d_color_palette[i])
			{
				d_result[ix] = i;
				break;
			}
}
//CLEANS GPU PALETTE FROM ZEROES
void cleanPalette(std::vector<int> &colors)
{
	auto vec = std::unique(colors.begin(), colors.end());
	colors.erase(vec, colors.end());
	if (colors.end() == colors.begin());
	colors.resize(colors.size() - 1);
}
void printIntPaletteToRgb(std::vector<int>& colors)
{
	int count = 0;
	for (auto v : colors)
	{
		int mask = 255;
		int r = v & mask;
		int g = (v & (mask << 8)) >> 8;
		int b = (v & (mask << 16)) >> 16;
		std::cout << count << ". ";
		std::cout << r << " ";
		std::cout << g << " ";
		std::cout << b << "\n";
		count++;
	}
}
//INPUT: BMP CLASS OBJECTS
//CALL GPU KERNELS :
//1. d_createColorPalette - Creating colors palette
//2. d_applyPalette - Applyes palette to resize the BMP file from 24 bit to 8 bit
//RUNS TEST TO VERIFY RESULT
void gpuCall(BMP img)
{
	//-------------------------------------------------------------------------------------------------------
	//GPU CREATE PALETTE
	//-------------------------------------------------------------------------------------------------------
	std::vector<int> all_colors_int(img.h_all_colors.size());
	for (int i = 0; i < img.h_all_colors.size(); i++)
		all_colors_int[i] = img.h_all_colors[i].convertRGBtoINT();
	int* d_all_colors;
	int* d_color_palette;
	std::vector<int> h_palette_from_gpu(256);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int* temp_colors;
	int* num;
	hipMalloc(&temp_colors, 10000 * sizeof(int));
	hipMalloc(&d_all_colors, all_colors_int.size() * sizeof(int));
	hipMalloc(&d_color_palette, 256 * sizeof(int));
	hipMalloc(&num, 1 * sizeof(int));
	//hipMemcpyToSymbol(HIP_SYMBOL(color_palette), img.h_color_palette.data(), 256);
	hipEventRecord(start);
	hipMemcpy(d_all_colors, all_colors_int.data(), all_colors_int.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_color_palette, h_palette_from_gpu.data(), 256 * sizeof(int), hipMemcpyHostToDevice);



	dim3 dimGrid(ceil(double(all_colors_int.size()) / 32.));
	dim3 dimBlock(32);
	d_createColorPalette<<<dimGrid, dimBlock>>> (d_all_colors, d_color_palette, all_colors_int.size(), temp_colors);
	hipDeviceSynchronize();
	

	dim3 gridWrite(ceil(10000. / 32.));
	dim3 blockWrite(32);
	writePalette <<< gridWrite, blockWrite >>> (d_color_palette, all_colors_int.size(), temp_colors, num);
	hipDeviceSynchronize();



	hipMemcpy(h_palette_from_gpu.data(), d_color_palette, 256 * sizeof(int), hipMemcpyDeviceToHost);
	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess) printf("%s ",hipGetErrorString(err1));
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	//cleanPalette(h_palette_from_gpu);
	std::cout << "GPU PALETTE:\n";
	printIntPaletteToRgb(h_palette_from_gpu);
	std::cout << "\ngpu milliseconds elapsed for creating palette: " << milliseconds << '\n';
	hipEventDestroy(start);
	hipEventDestroy(stop);
	//-------------------------------------------------------------------------------------------------------
	// APPLY PALETTE
	//-------------------------------------------------------------------------------------------------------
	std::vector<UINT8> h_applyPalette_result(all_colors_int.size());
	UINT8* d_applyPalette_result;
	hipEvent_t start1, stop1;
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipEventRecord(start1);
	hipMalloc(&d_applyPalette_result, all_colors_int.size() * sizeof(UINT8));
	dim3 Grid(ceil(double(all_colors_int.size())/32.));
	dim3 Block(32);
	d_applyPalette<<<Grid, Block>>> (d_all_colors, d_color_palette, d_applyPalette_result, all_colors_int.size());
	hipDeviceSynchronize();
	hipMemcpy(h_applyPalette_result.data(), d_applyPalette_result, all_colors_int.size() * sizeof(UINT8), hipMemcpyDeviceToHost);
	err1 = hipGetLastError();
	if (err1 != hipSuccess) printf("%s ", hipGetErrorString(err1));
	hipEventRecord(stop1);
	hipEventSynchronize(stop1);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start1, stop1);
	std::cout << "gpu milliseconds elapsed for applying palette: " << milliseconds << '\n';
	hipFree(d_all_colors);
	hipFree(d_color_palette);
	hipFree(d_applyPalette_result);


	//TEST decode on CPU and verify with origin
	BMP test;
	test.h_color_palette.resize(256);
	for (int i=0;i<h_palette_from_gpu.size(); i++)
	{
		test.h_color_palette[i] = BMP::RGB::convertINTtoRGB(h_palette_from_gpu[i]);
	}

	test.h_all_colors_resize.resize(h_applyPalette_result.size());
	test.h_all_colors_resize = h_applyPalette_result;
	test.returnColors(test.h_all_colors);
	bool check = true;
	for (int i = 0; i < all_colors_int.size(); i++)
		if (test.h_all_colors[i] != img.h_all_colors[i])
		{
			check = false;
			std::cout << i << " ";
			std::cout << (int)test.h_all_colors[i].red <<" " << (int)test.h_all_colors[i].green << " " << (int)test.h_all_colors[i].blue<<" and ";
			std::cout << (int)img.h_all_colors[i].red << " " << (int)img.h_all_colors[i].green << " " << (int)img.h_all_colors[i].blue;
			break;
		}
	std::cout << "TEST. create and apply palette on GPU and decode on CPU: ";
	if (check)
		std::cout << "ok";
	else std::cout << "not ok";
	std::cout << "\n";
}

int main()
{
    
	//printDeviceProp();
	//BMP image("testbmp.bmp");
	//BMP image("testbmp3.bmp");
	BMP image("parrotsscale.bmp");
	//BMP image("parrots.bmp");
	//BMP image("RAKETA.bmp");
	//BMP image("1556708032_1.bmp");
	image.collectAllColors();
	image.h_createColorPallete();
	image.h_applyPalette();

	std::cout << "CPU PALETTE:\n";
   image.printColorPallete();
	gpuCall(image);
	std::cout << "cpu milliseconds elapsed for creating palette: " << image.elapsed_palette.count() << "\n";
	std::cout << "cpu milliseconds elapsed for applying palette: " << image.elapsed_applying.count() << "\n";
}
